#include<iostream>
#include<hip/hip_runtime.h>
#include<math.h>
__global__ void grayscale(unsigned char* Pin ,unsigned char* Pout,int width, int height){
    int col=blockIdx.x*blockDim.x+threadIdx.x;
    int row=blockIdx.y*blockDim.y+threadIdx.y;
    
    if((row<height) && (col<width)){
        
        int idx=row*width+col;
        int ch_idx=idx*3;
        unsigned char r=Pin[ch_idx];
        unsigned char g=Pin[ch_idx+1];
        unsigned char b=Pin[ch_idx+2];
        Pout[idx]=0.21f*r+0.71f*g+0.07f*b;
        

    }
    
    return;
}

int main(){
    unsigned char hPin[10][10][3] = {
        { {1,2,3}, {4,5,6}, {7,8,9}, {10,11,12}, {13,14,15}, {16,17,18}, {19,20,21}, {22,23,24}, {25,26,27}, {28,29,30} },
        { {31,32,33}, {34,35,36}, {37,38,39}, {40,41,42}, {43,44,45}, {46,47,48}, {49,50,51}, {52,53,54}, {55,56,57}, {58,59,60} },
        { {61,62,63}, {64,65,66}, {67,68,69}, {70,71,72}, {73,74,75}, {76,77,78}, {79,80,81}, {82,83,84}, {85,86,87}, {88,89,90} },
        { {91,92,93}, {94,95,96}, {97,98,99}, {100,101,102}, {103,104,105}, {106,107,108}, {109,110,111}, {112,113,114}, {115,116,117}, {118,119,120} },
        { {121,122,123}, {124,125,126}, {127,128,129}, {130,131,132}, {133,134,135}, {136,137,138}, {139,140,141}, {142,143,144}, {145,146,147}, {148,149,150} },
        { {151,152,153}, {154,155,156}, {157,158,159}, {160,161,162}, {163,164,165}, {166,167,168}, {169,170,171}, {172,173,174}, {175,176,177}, {178,179,180} },
        { {181,182,183}, {184,185,186}, {187,188,189}, {190,191,192}, {193,194,195}, {196,197,198}, {199,200,201}, {202,203,204}, {205,206,207}, {208,209,210} },
        { {211,212,213}, {214,215,216}, {217,218,219}, {220,221,222}, {223,224,225}, {226,227,228}, {229,230,231}, {232,233,234}, {235,236,237}, {238,239,240} },
        { {241,242,243}, {244,245,246}, {247,248,249}, {250,251,252}, {253,254,255}, {0,1,2}, {3,4,5}, {6,7,8}, {9,10,11}, {12,13,14} },
        { {15,16,17}, {18,19,20}, {21,22,23}, {24,25,26}, {27,28,29}, {30,31,32}, {33,34,35}, {36,37,38}, {39,40,41}, {42,43,44} }
    };

    unsigned char  *dPout, *dPin , *hPout;
    hPout=(unsigned char *)malloc(sizeof(hPin)/3);
    int height,width;
    height=sizeof(hPin)/sizeof(hPin[0]);
    width=sizeof(hPin[0])/3;

    hipMalloc((void**) &dPout, sizeof(hPin)/3);
    hipMalloc((void**) &dPin, sizeof(hPin));

    hipMemcpy(dPin,hPin,sizeof(hPin),hipMemcpyHostToDevice);
    dim3 grid(ceil(width/16.0f),ceil(height/16.0f),1);
    dim3 block(16,16,1);

  
    grayscale<<<grid,block>>> (dPin,dPout,width,height);
    hipMemcpy(hPout,dPout,sizeof(hPin)/3,hipMemcpyDeviceToHost);
    std::cout<< (int)hPout[55];
    return 0 ;
}
