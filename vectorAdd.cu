#include <iostream>
#include <hip/hip_runtime.h>

__global__
void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}


int main(){
    float *h_A,*h_B,*h_C,*d_A,*d_B,*d_C;
    int n=2;
    float h_a[2]={1.0,2.0};
    size_t size = n * sizeof(float);
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(size);
    for (int i=0 ; i<n;i++){
        h_A[i]=2.0;
        h_B[i]=1.0;
    }
    
    hipMalloc((void**) &d_A,size);
    hipMalloc((void**) &d_B,size);
    hipMalloc((void**) &d_C,size);
    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
    vectorAdd <<< (n + 255) / 256, 256 >>> (d_A, d_B, d_C, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
    hipDeviceSynchronize();
    hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
    for (int i=0 ; i<n;i++){
        std::cout<<h_C[i]<<std::endl;
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}